#include "hip/hip_runtime.h"
#include "adNOC_2Pb_runKernels.cuh"
#include <assert.h>

RunKernels::RunKernels(char *fname)
{
    readinParamFile(fname);
    
    dataInitialization();
}

void RunKernels::readinParamFile(char *fname)
{
	FILE* FID =  fopen(fname, "r");
	if(!FID) {
		printf("Parameter file %s is missing. Abort...\n", fname);
		exit(1);
	}

	char line[1024];

	readParamLine(FID, line, 1024);
	sscanf(line, "%lf", &TotalSim);
  
	readParamLine(FID, line, 1024);
	sscanf(line, "%lf", &EachOut);
	
	readParamLine(FID, line, 1024);
	sscanf(line, "%lf", &delta0);
	
	readParamLine(FID, line, 1024);
	sscanf(line, "%lf", &Cd);
	
	readParamLine(FID, line, 1024);
	sscanf(line, "%lf", &N_R);
	
	readParamLine(FID, line, 1024);
	sscanf(line, "%lf", &varTheta);
	
	readParamLine(FID, line, 1024);
	sscanf(line, "%lf", &phiS0);

	readParamLine(FID, line, 1024);
	sscanf(line, "%s", DEMData);
	
	readParamLine(FID, line, 1024);
	sscanf(line, "%s", IniHData);

	readParamLine(FID, line, 1024);
	sscanf(line, "%s", IniUData);

	readParamLine(FID, line, 1024);
	sscanf(line, "%s", IniVData);

	readParamLine(FID, line, 1024);
	sscanf(line, "%s", locData);
	
}

void RunKernels::readParamLine(FILE *fid, char *line, int len)
{
	char *c;
	fgets(line, len, fid);

	// Remove the things after '#', or end of line ('\n')
	for(c = line; *c; c++) {
		int br = 0;
		switch(*c) {
		case '#':
		case '\n':
		*c = 0;
		br = 1;
		break;
		default:
		break;
		}
		if(br) break;
	}

	return;
}

void RunKernels::split(const string& s, vector<string>& sv, const char* delim)
{
	sv.clear();
	char* buffer = new char[s.size() + 1];
	buffer[s.size()] = '\0';
	copy(s.begin(), s.end(), buffer);

	char* p = std::strtok(buffer, delim);
	do {
		sv.push_back(p);
	} while ((p = strtok(NULL, delim)));

	delete[] buffer;

	return;
}

void RunKernels::dataInitialization()
{
	fprintf(stdout,"\n\t***********************************************************************************\n");
	fprintf(stdout,"\t\t* * * * *     MoSES_2PDF:  inflow (debris flow) mode     * * * * * \n");
	fprintf(stdout,"\t-----------------------------------------------------------------------------------\n");
  	fprintf(stdout,"\t\t2D Central Scheme  (adNOC) Mixture Code in CUDA: inflow mode\n");
	fprintf(stdout,"\t\tAuthor : Chi-Jyun Ko, Po-Chih Chen, Hock-Kiet Wong and Yih-Chin Tai\n");
	fprintf(stdout,"\t\tLab for Computer Simulation and Visualization (CSV Lab), NCKU, Taiwan\n");  
  	fprintf(stdout,"\t***********************************************************************************\n\n");
	

	ifstream inputFileTopo;
	ifstream inputFileInitH;
	ifstream inputFileInitU;
	ifstream inputFileInitV;
	ifstream inputFileloc;
  
	inputFileTopo.open(DEMData,  ios::in);
	inputFileInitH.open(IniHData, ios::in);
	inputFileInitU.open(IniUData, ios::in);
	inputFileInitV.open(IniVData, ios::in);
	inputFileloc.open(locData, ios::in);

	fprintf(stdout,"\t\tinput Topo File        : %s\n", DEMData);
	fprintf(stdout,"\t\tinput Initial File (H) : %s\n", IniHData);
	fprintf(stdout,"\t\tinput Initial File (U) : %s\n", IniUData);
	fprintf(stdout,"\t\tinput Initial File (V) : %s\n\n", IniVData);


	// read Topo data
	if (inputFileTopo.fail() ){
	 	printf("\n\t----------------------------------\n");
		printf("\t   Error can't open Topo file.\n");
		printf("\t----------------------------------\n");
		assert(0);
	}

	// read init data
	if (inputFileInitH.fail() ){
		printf("\n\t-------------------------------------------------\n");
		printf("\t   Error can't open Initial depth file.\n");
		printf("\t-------------------------------------------------\n");
		assert(0);
	}

	if (inputFileInitU.fail() ){
		printf("\n\t-------------------------------------------------\n");
		printf("\t   Error can't open Initial velocity(U) file.\n");
		printf("\t-------------------------------------------------\n");
		assert(0);
	}

	if (inputFileInitV.fail() ){
		printf("\n\t-------------------------------------------------\n");
		printf("\t   Error can't open Initial velocity(V) file.\n");
		printf("\t-------------------------------------------------\n");
		assert(0);
	}

	if (inputFileloc.fail() ){
		printf("\n\t-------------------------------------------------\n");
		printf("\t   Error can't open Inflow location file.\n");
		printf("\t-------------------------------------------------\n");
		assert(0);
	}

	
	while(getline(inputFileTopo,TopoS)){
		inputTopoTmp.push_back(TopoS);
	}

	getline(inputFileInitH, IniHS);
	while(getline(inputFileInitH,IniHS)){
		inputIniHTmp.push_back(IniHS);
		
	}

	getline(inputFileInitU, IniUS);
	while(getline(inputFileInitU,IniUS)){
		inputIniUTmp.push_back(IniUS);
	}

	getline(inputFileInitV, IniVS);
	while(getline(inputFileInitV,IniVS)){
		inputIniVTmp.push_back(IniVS);
	}

	getline(inputFileloc, locS);
	while(getline(inputFileloc,locS)){
		inputlocTmp.push_back(locS);
	}

	// read Topo file
	for(int i=0;i<6;i++){
		split(inputTopoTmp[i], Topodata, " ");
		if(i==0){
			NX = stoi(Topodata[1]);
		}
		else if(i==1){
			NY = stoi(Topodata[1]);
		}
		else if(i==2){
			xllcorner = stof(Topodata[1]);
		}else if(i==3){
			yllcorner = stof(Topodata[1]);
		}
		else if(i==4){
			dx = stof(Topodata[1]);
			dy = stof(Topodata[1]);
		}
		else if(i==5){
			if(Topodata[0] == "NODATA_value"){
				StartTopo = 0;
			}
			else{
				StartTopo = 1;
			}
		}
	}


	Iniflowlen = inputIniHTmp.size();
	locflowlen = inputlocTmp.size();

	nx = NX;
	ny = NY;

	dx = dx*10;
	dy = dy*10;

	MINX = 0.0;
	MINY = 0.0;

	MAXX = dx*(nx-1);
	MAXY = dy*(ny-1);

	nxd = nx + 2*MD;
	nyd = ny + 2*MD;

	nxyd = max(nxd,nyd);

	arraySize = nxd * nyd;

	fprintf(stdout,"\t\tData points          : %d,%d\n", nx, ny);
  	fprintf(stdout,"\t\tDomain [dm]          : (%5.3f,%5.3f)(%5.3f,%5.3f)\n",
		MINX, MAXX, MINY, MAXY);
  	fprintf(stdout,"\t\tGrid size            : %6.2f,%6.2f (%d,%d)\n",dx,dy,nx,ny);
  	fprintf(stdout,"\t\tCFL number           : %5.3f\n", CFL);
	fprintf(stdout,"\t\tdelta0               : %5.3f\n", delta0);
	fprintf(stdout,"\t\tCd                   : %5.3f\n", Cd);
	fprintf(stdout,"\t\tN_R                  : %5.3f\n", N_R);
	fprintf(stdout,"\t\tvarTheta             : %5.3f\n", varTheta);
	fprintf(stdout,"\t\tinitial value of solid volume fraction : %5.3f\n\n", phiS0);

	fprintf(stdout,"\t\tTotal simulation time (sec) : %5.3f\n", TotalSim);
	fprintf(stdout,"\t\tEach output time (sec)      : %5.3f\n\n", EachOut);
	  
	NEW_MATRIX(topo, double, nxd, nyd);
	NEW_MATRIX(depth, double, nxd, nyd);

	NEW_MATRIX(inputLoc,    double, 3, locflowlen);
	NEW_3DMATRIX(inputFlow, double, locflowlen, Iniflowlen, 3);
	NEW_ARRAY(inflowTime, double, Iniflowlen);

	NEW_MATRIX(dire,    int, 3, locflowlen);

	// input Topo to matrix
	if(StartTopo==1){
		for(int j=0; j<NY; j++) {
			split(inputTopoTmp[j+5], Topodata, " ");
			
			for(int i=0; i<NX; i++) {
				topo[i+MD][j+MD] = stof(Topodata[i]);
				topo[i+MD][j+MD] = 10.0*topo[i+MD][j+MD];

				if(topo[i+MD][j+MD]<0){
					topo[i+MD][j+MD] = 0;
				}
			}
	
		}
	}
	else{
		for(int j=0; j<NY; j++) {
			split(inputTopoTmp[j+6], Topodata, " ");
			
			for(int i=0; i<NX; i++) {
				topo[i+MD][j+MD] = stof(Topodata[i]);
				topo[i+MD][j+MD] = 10.0*topo[i+MD][j+MD];

				if(topo[i+MD][j+MD]<0){
					topo[i+MD][j+MD] = 0;
				}
			}
	
		}
	}

	// input Initial flow
	inflowSize = Iniflowlen*locflowlen;

	for(int j=0; j<Iniflowlen; j++) {
		
		split(inputIniHTmp[j], IniHdata, "\t");
		split(inputIniUTmp[j], IniUdata, "\t");
		split(inputIniVTmp[j], IniVdata, "\t");

		inflowTime[j] = stof(IniHdata[0])*10;

		for(int i=0; i<locflowlen; i++) {

			inputFlow[i][j][0] = 10.0*stof(IniHdata[i+1]);
			inputFlow[i][j][1] = stof(IniUdata[i+1]);
			inputFlow[i][j][2] = stof(IniVdata[i+1]);


		}

	}

	
	
	
	// input Initial flow location
	for(int j=0; j<locflowlen; j++) {
		split(inputlocTmp[j], locdata, "\t");
		// cout <<stof(IniHdata[0]) <<" ";
		for(int i=0; i<3; i++) {
			inputLoc[i][j] = stof(locdata[i]);
			// cout<<inputLoc[i][j] <<" ";
		}
		// cout <<"\n";
	}
	

	// B.C. initioal condition
	for(int i=0; i<MD; i++) {
		for(int j=MD; j<(ny+MD); j++) {
			topo[i      ][j] = topo[MD     ][j];
			topo[nx+MD+i][j] = topo[nx+MD-1][j];
		
		}
	}

	for(int j=0; j<MD; j++) {
		for(int i=0; i<(nx+2*MD); i++) {
			topo[i][j      ] = topo[i][MD     ];
			topo[i][ny+MD+j] = topo[i][ny+MD-1];

		}
	}
	

}

clock_t RunKernels::run()
{
	hipError_t errhost ;
	errhost = hipHostMalloc((void **)&TotalStep_h, sizeof(double));
	errhost = hipHostMalloc((void **)&dt_h, sizeof(double));

	errhost = hipHostMalloc((void **)&depth_h, sizeof(double) * arraySize);
	errhost = hipHostMalloc((void **)&topo_h, sizeof(double) * arraySize);
	errhost = hipHostMalloc((void **)&speed_h, sizeof(double) * arraySize);

	errhost = hipHostMalloc((void **)&resultHs_h, sizeof(double) * arraySize);
	errhost = hipHostMalloc((void **)&resultHf_h, sizeof(double) * arraySize);
	errhost = hipHostMalloc((void **)&resultUs_h, sizeof(double) * arraySize);
	errhost = hipHostMalloc((void **)&resultVs_h, sizeof(double) * arraySize);
	errhost = hipHostMalloc((void **)&resultUf_h, sizeof(double) * arraySize);
	errhost = hipHostMalloc((void **)&resultVf_h, sizeof(double) * arraySize);

	errhost = hipHostMalloc((void **)&resultphi_h, sizeof(double) * arraySize);
	
	errhost = hipHostMalloc((void **)&bfkt_h,   sizeof(double) * arraySize * 3);
	errhost = hipHostMalloc((void **)&svec_h,   sizeof(double) * arraySize * 2);
	errhost = hipHostMalloc((void **)&cvalue_h, sizeof(double) * arraySize);
	errhost = hipHostMalloc((void **)&result_h, sizeof(double) * arraySize);

	errhost = hipHostMalloc((void **)&inflow_h, sizeof(double) * inflowSize * 3);
	errhost = hipHostMalloc((void **)&loc_h, sizeof(double) * locflowlen * 3);
	errhost = hipHostMalloc((void **)&dire_h, sizeof(int) * locflowlen * 3);

	if(errhost != hipSuccess){
		printf("\nCould not allocate Host memory : %d\n",errhost);
	}
	
	for(int i = 0; i < nxd; i++){
		for(int j = 0; j < nyd; j++){
		
			topo_h[j*nxd+i] =  topo[i][j];
		// depth_h[j*nxd+i] = depth[i][j];

		}
	}

	for(int m=0;m<3;m++){
		for(int j = 0; j < Iniflowlen; j++){
			for(int i = 0; i < locflowlen; i++){
				
				inflow_h[m * Iniflowlen * locflowlen + j*locflowlen + i] =  inputFlow[i][j][m];
			
			}
		}
	}

	for(int j=0;j<locflowlen;j++){
		for(int i=0;i<3;i++){
			if((inputLoc[2][j] == 0) || (inputLoc[2][j] == NY)){
				dire[0][j] =  inputLoc[0][j];
				dire[1][j] = -5; //dire[0] direction in X
				dire[2][j] =  0; //dire[1] direction in Y
			}else{
				dire[0][j] = inputLoc[0][j]; 
				dire[1][j] = 0; 
				dire[2][j] = -5;
			}
			// cout<<dire[i][j]<<" ";
		}
		// cout<<endl;
	}

	for(int i = 0; i < 3; i++){
		for(int j = 0; j < locflowlen; j++){			
			 loc_h[j*3+i] =  inputLoc[i][j];
			dire_h[j*3+i] =  dire[i][j];	
		}
	}

	clock_t start, end;

	start = clock(); //cuda start
	memoryMalloc();
	
	kernelStep();
	
	freeMemory();
	end = clock(); //cuda stop

	// outputFile();
	
	return end - start;
}

void RunKernels::kernelStep()
{

	hipError_t errMem, errCpy ;

	errMem = hipMemset(dev_TotalTime, 0.0, sizeof(double));
	errMem = hipMemset(dev_dt, 0.0, sizeof(double));
	errMem = hipMemset(dev_dtval, 0.0, sizeof(double));
	errMem = hipMemset(dt_h, 0.0, sizeof(double));

	if(errMem != hipSuccess){
		printf("\nError cuda Memory set : %d\n",errMem);
	}
	
	errCpy = hipMemcpy(dev_topo,   topo_h, sizeof(double) * arraySize, hipMemcpyHostToDevice);
	errCpy = hipMemcpy(dev_inflow,   inflow_h, sizeof(double) * (inflowSize * 3), hipMemcpyHostToDevice);
	errCpy = hipMemcpy(dev_loc,      loc_h,    sizeof(double) * (locflowlen * 3), hipMemcpyHostToDevice);
	errCpy = hipMemcpy(dev_dire,     dire_h,   sizeof(int)    * (locflowlen * 3), hipMemcpyHostToDevice);
	

	int bx = (nxd + BLOCK_SIZE - 1) / BLOCK_SIZE;
	int by = (nyd  + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 blocksPerGrid(bx, by);
	dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);

	double Htmp, hstmp, hftmp, ustmp, uftmp, vstmp, vftmp, phitmp;

	int outputStep = TotalSim/EachOut;
	double tf[outputStep+1] = {0.0};

	for(int ii=1;ii<=outputStep;ii++){
		tf[ii] = ii*EachOut*10;
	}

	double outtime[outputStep+1]={0};
	int iter = 1, nt, io;
	int nstop = 0, schreiben =0;
	// int Totalnt = 0;
	int outsteplen = sizeof(tf)/sizeof(tf)[0];
	int inflowCount =0;

	makeTopo1Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_bfkt,
		MINX, MINY,
		dx, dy,
		nxd, nyd);
	hipDeviceSynchronize();

	makeTopo2Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_topo,
		dev_bfkt,
		dx, dy,
		nxd, nyd, nx, ny);
	hipDeviceSynchronize();

	makeTopo3Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_bfkt,
		dev_posx, dev_posy,
		nxd, nyd);
	hipDeviceSynchronize();

	makeTopo4Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_bfkt,
		dev_posx, dev_posy,
		dev_dxdxi11, dev_dxdxi12,
		dev_dxdxi21, dev_dxdxi22,
		dev_dbdx, dev_dbdy,
		nxd, nyd);
	hipDeviceSynchronize();

	makeTopo5Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_dbdx, dev_dbdy,
		dev_cvalue,
		nxd, nyd);
	hipDeviceSynchronize();

	makeTopo6Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_dbdx, dev_dbdy,
		dev_cvalue,
		dev_svec,
		nxd, nyd);
	hipDeviceSynchronize();

	makeTopo7Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_dbdx, dev_dbdy,
		dev_cvalue,
		dev_svec,
		dev_Jacb31, dev_Jacb32,
		dev_dxdxi11, dev_dxdxi12, 
		dev_dxdxi21, dev_dxdxi22,
		dev_dettmp,  
		nxd, nyd);
	hipDeviceSynchronize();

	makeTopo8Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_cvalue, dev_svec,
		dev_Jacb31, dev_Jacb32,
		dev_dxdxi11, dev_dxdxi12, 
		dev_dxdxi21, dev_dxdxi22,
		dev_dettmp,  
		dev_Detmin,
		dev_i_ddxi11, dev_i_ddxi12, 
		dev_i_ddxi21, dev_i_ddxi22, 
		nxd, nyd);
	hipDeviceSynchronize();

	makeTopo9Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_cvalue, dev_svec,
		dev_i_ddxi11, dev_i_ddxi12, 
		dev_i_ddxi21, dev_i_ddxi22,
		dev_invJ11, dev_invJ12, dev_invJ13,  
		dev_invJ21, dev_invJ22, dev_invJ23,
		dev_invJ31, dev_invJ32, dev_invJ33,
		nxd, nyd);
	hipDeviceSynchronize();

	makeTopo11Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_u,
		dev_tande, delta0,
		nxd, nyd);
	hipDeviceSynchronize();

	Boundary1Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_dxdxi11, dev_dxdxi12,
		dev_dxdxi21, dev_dxdxi22,
		dev_cvalue, dev_Detmin,
		dev_svec,
		dev_Jacb31, dev_Jacb32,
		dev_invJ11, dev_invJ12, dev_invJ13,
		dev_invJ21, dev_invJ22, dev_invJ23,
		dev_invJ31, dev_invJ32, dev_invJ33,  
		nxd, nyd, nx ,ny);
	hipDeviceSynchronize();

	Boundary2Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_dxdxi11, dev_dxdxi12,
		dev_dxdxi21, dev_dxdxi22,
		dev_cvalue, dev_Detmin,
		dev_svec,
		dev_Jacb31, dev_Jacb32,
		dev_invJ11, dev_invJ12, dev_invJ13,
		dev_invJ21, dev_invJ22, dev_invJ23,
		dev_invJ31, dev_invJ32, dev_invJ33,  
		nxd, nyd, nx ,ny);
	hipDeviceSynchronize();

	JacobKernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_svec, dev_cvalue,
		dev_posx, dev_posy, 
		dev_J13dxi, dev_J23dxi, dev_J33dxi, 
		dev_J13det, dev_J23det, dev_J33det,  
		nxd, nyd);
	hipDeviceSynchronize();

	Boundary3Kernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_J13dxi, dev_J23dxi, dev_J33dxi, 
		dev_J13det, dev_J23det, dev_J33det,  
		nxd, nyd, nx ,ny);
	hipDeviceSynchronize();


	errCpy = hipMemcpy(bfkt_h,   &dev_bfkt[0],   sizeof(double)* arraySize * 3, hipMemcpyDeviceToHost);
	errCpy = hipMemcpy(svec_h,   &dev_svec[0],   sizeof(double)* arraySize * 2, hipMemcpyDeviceToHost);
	errCpy = hipMemcpy(cvalue_h, &dev_cvalue[0], sizeof(double)* arraySize, hipMemcpyDeviceToHost);
				
	FILE  *fpTopo;

	if ((fpTopo=fopen("./result2Pb/DEM.dat", "w")) == NULL)
    {
		printf("\n\t---------------------------------------------------------\n");
		printf("\t  Error can't open \"result2Pb\" folder.\n");
		printf("\t  Need build the result directory  --> mkdir result2Pb\n");
		printf("\t---------------------------------------------------------\n");
		
		fclose(fpTopo);
		exit(0);
    }
	fprintf(fpTopo, "VARIABLES = \"x\", \"y\", \"z\", \"c\", \"S1\", \"S2\"\n ");
	for (int i=MD;i<nxd-MD;i++) {
	  for (int j=MD;j<nyd-MD;j++) {
		fprintf(fpTopo, "%10.4f\t%10.4f\t%10.4f\t%10.4f\t%10.4f\t%10.4f\n",bfkt_h[0 * nxd * nyd + j * nxd + i]*0.1, bfkt_h[1 * nxd * nyd + j * nxd + i]*0.1, bfkt_h[2 * nxd * nyd + j * nxd + i]*0.1, cvalue_h[j * nxd + i],svec_h[0 * nxd * nyd + j * nxd + i],svec_h[1 * nxd * nyd + j * nxd + i]);
	  } 
	}	
	fclose(fpTopo);

	FILE  *fpInit;

	if ((fpInit=fopen("./result2Pb/001.dat", "w")) == NULL)
    {
		printf("\n\t---------------------------------------------------------\n");
		printf("\t  Error can't open \"result2Pb\" folder.\n");
		printf("\t---------------------------------------------------------\n");
		fclose(fpInit);
		exit(0);
    }
	fprintf(fpInit, "VARIABLES = \"H\", \"phi\", \"Us\", \"Uf\", \"Vs\", \"Vf\"\n ");
	for (int i=MD;i<nxd-MD;i++) {
		for (int j=MD;j<nyd-MD;j++) {
			fprintf(fpInit, "%10.4f\t%10.4f\t%10.4f\t%10.4f\t%10.4f\t%10.4f\n",0.1*depth[i][j],phiS0, 0.0, 0.0, 0.0, 0.0);
		} 
	}
	fclose(fpInit);

	FILE  *fpInfo;
	if ((fpInfo=fopen("./result2Pb/Info.dat", "w")) == NULL)
    {
		printf("\n\t---------------------------------------------------------\n");
		printf("\t  Error can't open \"result2Pb\" folder.\n");
		printf("\t---------------------------------------------------------\n");
    	fclose(fpInfo);
    	exit(0);
	}
	fprintf(fpInfo, "VARIABLES = \"x-point\", \"y-point\", \"dx\", \"dy\", \"xllcorner\", \"yllcorner\", \"TotalStep\"\n ");
	fprintf(fpInfo, "\t%d\t\t %d\t %10.2f\t %10.2f\t %10.4f\t %10.4f\t %d", NX, NY, (dx*0.1), (dy*0.1), xllcorner, yllcorner,(outputStep+1));
	fclose(fpInfo);

	hipFree(dev_i_ddxi11);
	hipFree(dev_i_ddxi12);
	hipFree(dev_i_ddxi21);
	hipFree(dev_i_ddxi22);

	hipFree(dev_dettmp);
	hipFree(dev_bfkt);

	MeanKernel <<<blocksPerGrid, threadsPerBlock>>>(
		dev_result, 
		dev_dxdxi11, dev_dxdxi21,
		dev_dxdxi12, dev_dxdxi22,

		dev_J13dxi, dev_J23dxi, dev_J33dxi, 
		dev_J13det, dev_J23det, dev_J33det, 
		
		dev_invJ11, dev_invJ12, dev_invJ13, 
		dev_invJ21, dev_invJ22, dev_invJ23, 
		dev_invJ31, dev_invJ32, dev_invJ33, 

		dev_Detmin, dev_cvalue, dev_svec, 

		dev_dxdxi11_avgEW, dev_dxdxi21_avgEW,
		dev_dxdxi12_avgSN, dev_dxdxi22_avgSN, 
		
		dev_J13dxi_avgEW, dev_J23dxi_avgEW, dev_J33dxi_avgEW,
		dev_J13det_avgEW, dev_J23det_avgEW, dev_J33det_avgEW,

		dev_J13dxi_avgSN, dev_J23dxi_avgSN, dev_J33dxi_avgSN,
		dev_J13det_avgSN, dev_J23det_avgSN, dev_J33det_avgSN,

		dev_invJ11_avgEW, dev_invJ12_avgEW, dev_invJ13_avgEW,
		dev_invJ21_avgEW, dev_invJ22_avgEW, dev_invJ23_avgEW,
		dev_invJ31_avgEW, dev_invJ32_avgEW, dev_invJ33_avgEW,

		dev_invJ11_avgSN, dev_invJ12_avgSN, dev_invJ13_avgSN,
		dev_invJ21_avgSN, dev_invJ22_avgSN, dev_invJ23_avgSN,
		dev_invJ31_avgSN, dev_invJ32_avgSN, dev_invJ33_avgSN,

		dev_Detmin_avgEW, dev_Detmin_avgSN,
		dev_cval_avgEW,   dev_cval_avgSN,
		dev_svec_avgEW,   dev_svec_avgSN,  

		nxd, nyd);
	hipDeviceSynchronize();



	for (nt = 1; (!nstop) && (nt<100000); nt++){

		Inflow1Kernel <<<blocksPerGrid, threadsPerBlock>>>(
			dev_result, 
			dev_inflow, dev_loc, 
			dev_u, dev_cvalue,
			phiS0,
			locflowlen, Iniflowlen, inflowCount,
			dev_dire,
			nxd, nyd);
		hipDeviceSynchronize();

		Inflow2Kernel <<<blocksPerGrid, threadsPerBlock>>>(
			dev_result, 
			dev_inflow, dev_loc, 
			dev_u,
			locflowlen, Iniflowlen, inflowCount,
			dev_dire,
			nxd, nyd);
		hipDeviceSynchronize();

		for (io=0; io<2; io++){

			if(io == 0){
				UzeroKernel <<<blocksPerGrid, threadsPerBlock>>>(
					dev_result, 
					dev_u, dev_uzero, 
					nxd, nyd);
				hipDeviceSynchronize();
			}

			hipDeviceSynchronize();

			Boundary5Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_u,  
				nxd, nyd, nx ,ny);
			hipDeviceSynchronize();

			Boundary6Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_u,  
				nxd, nyd, nx ,ny);
			hipDeviceSynchronize();

			Boundary7Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_u,  
				nxd, nyd, nx ,ny);
			hipDeviceSynchronize();


			Boundary9Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_Hpx, dev_Hpy,
				dev_Ppx, dev_Ppy,
				dev_PDx, dev_PDy,
				dev_ux,  dev_uy,
				dev_apEW, dev_apSN,
				dev_apFEW, dev_apFSN,   
				nxd, nyd);
			hipDeviceSynchronize();

			TVD1Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_u, 
				dev_dux, dev_duy,
				nxd, nyd);
			hipDeviceSynchronize();

			TVD2Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_dux, dev_duy,
				dev_sgnAx, dev_sgnBx,
				dev_sgnAy, dev_sgnBy, 
				nxd, nyd);
			hipDeviceSynchronize();

			TVD3Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_dux, dev_duy,
				dev_sgnAx, dev_sgnBx,
				dev_sgnAy, dev_sgnBy, 
				dev_t1x, dev_t2x,
				dev_t1y, dev_t2y, 
				nxd, nyd);
			hipDeviceSynchronize();

			TVD4Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_t1x, dev_t2x,
				dev_t1y, dev_t2y,
				dev_sgnAx, dev_sgnBx,
				dev_sgnAy, dev_sgnBy, 
				nxd, nyd);
			hipDeviceSynchronize();

			TVD5Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_t1x, dev_t2x,
				dev_t1y, dev_t2y,
				dev_sgnAx, dev_sgnBx,
				dev_sgnAy, dev_sgnBy, 
				dev_ux, dev_uy, 
				nxd, nyd);
			hipDeviceSynchronize();

			InterfacesKernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_u, 
				dev_ux, dev_uy,
				dev_uE, dev_uW, 
				dev_uN, dev_uS,  
				nxd, nyd);
			hipDeviceSynchronize();

			Interfaces2Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_uE, dev_uW, 
				dev_uN, dev_uS,  
				nxd, nyd,nx ,ny);
			hipDeviceSynchronize();


			KeepPositivi1Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_uE, dev_uW, 
				dev_uN, dev_uS,  
				nxd, nyd);
			hipDeviceSynchronize();

			KeepPositivi2Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_uE, dev_uW, 
				dev_uN, dev_uS,  
				nxd, nyd);
			hipDeviceSynchronize();

			Flux1Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_uE, dev_uW, 
				dev_uN, dev_uS,  
				dev_vexE, dev_veyE, 
				dev_vexW, dev_veyW,
				dev_vexFE, dev_veyFE, 
				dev_vexFW, dev_veyFW,

				dev_vexN, dev_veyN, 
				dev_vexS, dev_veyS,
				dev_vexFN, dev_veyFN, 
				dev_vexFS, dev_veyFS,

				nxd, nyd);
			hipDeviceSynchronize();

			Flux2Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_uE, dev_uW, 
				dev_uN, dev_uS,  
				dev_vexE, dev_veyE, 
				dev_vexW, dev_veyW,
				dev_vexFE, dev_veyFE, 
				dev_vexFW, dev_veyFW,

				dev_vexN, dev_veyN, 
				dev_vexS, dev_veyS,
				dev_vexFN, dev_veyFN, 
				dev_vexFS, dev_veyFS,

				dev_w_wertE, dev_w_wertW,
				dev_w_wertFE, dev_w_wertFW,

				dev_w_wertN, dev_w_wertS,
				dev_w_wertFN, dev_w_wertFS,
				
				dev_svec, dev_cvalue,

				nxd, nyd);
			hipDeviceSynchronize();

			Flux3Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_uE, dev_uW, 
				dev_uN, dev_uS,  
				dev_vexE, dev_veyE, 
				dev_vexW, dev_veyW,
				dev_vexFE, dev_veyFE, 
				dev_vexFW, dev_veyFW,

				dev_vexN, dev_veyN, 
				dev_vexS, dev_veyS,
				dev_vexFN, dev_veyFN, 
				dev_vexFS, dev_veyFS,

				dev_w_wertE, dev_w_wertW,
				dev_w_wertFE, dev_w_wertFW,

				dev_w_wertN, dev_w_wertS,
				dev_w_wertFN, dev_w_wertFS,
				
				dev_q_xiE , dev_q_etE,
				dev_q_xiW , dev_q_etW,
				dev_q_xiFE, dev_q_etFE,
				dev_q_xiFW, dev_q_etFW,

				dev_NpressFE, dev_NpressFW, dev_M11EW,
				
				dev_invJ11_avgEW, dev_invJ12_avgEW, dev_invJ13_avgEW,
				dev_invJ21_avgEW, dev_invJ22_avgEW, dev_invJ23_avgEW, 
				dev_cval_avgEW,

				dev_q_xiN , dev_q_etN,
				dev_q_xiS , dev_q_etS,
				dev_q_xiFN, dev_q_etFN,
				dev_q_xiFS, dev_q_etFS,

				dev_NpressFN, dev_NpressFS, dev_M22SN,
				
				dev_invJ11_avgSN, dev_invJ12_avgSN, dev_invJ13_avgSN,
				dev_invJ21_avgSN, dev_invJ22_avgSN, dev_invJ23_avgSN, 
				dev_cval_avgSN,

				nxd, nyd);
			hipDeviceSynchronize();

			Flux4Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_uE, dev_uW, 
				dev_uN, dev_uS, 
				
				dev_q_xiE , dev_q_xiW,
				dev_q_xiFE, dev_q_xiFW,

				dev_NpressFE, dev_NpressFW, dev_invJ11_avgEW,
				
				dev_apE, dev_apW,
				dev_apFE, dev_apFW, 

				dev_q_etN , dev_q_etS,
				dev_q_etFN, dev_q_etFS,

				dev_NpressFN, dev_NpressFS, dev_invJ22_avgSN,

				dev_apN, dev_apS,
				dev_apFN, dev_apFS, 

				nxd, nyd);
			hipDeviceSynchronize();

			Flux5Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_apE, dev_apW,
				dev_apFE, dev_apFW, 

				dev_apEW , dev_apFEW,

				dev_apN, dev_apS,
				dev_apFN, dev_apFS, 

				dev_apSN, dev_apFSN,

				nxd, nyd);
			hipDeviceSynchronize();


			Flux6Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_apEW, dev_apFEW,
				dev_apSN, dev_apFSN,

				dev_em_x ,  dev_em_y,
				dev_em_Fx, dev_em_Fy,

				dev_czw1x, dev_czw2x,
				dev_czwF1x,dev_czwF2x, 
				dev_czw1y, dev_czw2y,
				dev_czwF1y,dev_czwF2y,
				
				dev_uE, dev_uW, 
				dev_uN, dev_uS,
				
				dev_cval_avgEW, dev_cval_avgSN,
				dev_Detmin_avgEW, dev_Detmin_avgSN,
				dev_M11EW, dev_M22SN,

				nxd, nyd);
			hipDeviceSynchronize();

			Flux7Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_FpE, dev_FpW,
				dev_GpN, dev_GpS,

				dev_czw1x, dev_czw2x,
				dev_czwF1x,dev_czwF2x, 
				dev_czw1y, dev_czw2y,
				dev_czwF1y,dev_czwF2y,
				
				dev_uE, dev_uW, 
				dev_uN, dev_uS,
				
				dev_Detmin_avgEW, dev_Detmin_avgSN,
				
				dev_q_xiE, dev_q_xiFE,
				dev_q_xiW, dev_q_xiFW,

				dev_q_etN, dev_q_etFN,
				dev_q_etS, dev_q_etFS,

				dev_dxdxi11_avgEW, dev_dxdxi21_avgEW,  
				dev_dxdxi12_avgSN, dev_dxdxi22_avgSN, 

				nxd, nyd);
			hipDeviceSynchronize();


			CFL1Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_em_x, dev_em_y,
				dev_em_Fx, dev_em_Fy,
				dev_em_valS, dev_em_valF,
				dx, dy,
				nxd, nyd);
			hipDeviceSynchronize();

			CFL2Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_em_valS, dev_em_valF,
				dev_Val,
				nxd, nyd);
			hipDeviceSynchronize();
			
			if(io==0){
			
				int threads = 256;
				int blocks = 256;//min((arraySize + threads - 1) / threads, 256);
				
				reduceKernel <<<blocks, threads>>> (dev_Val, dev_max, arraySize);
				hipDeviceSynchronize();

				reduceKernel <<<1, blocks>>> (dev_max, dev_maxW, blocks);
				hipDeviceSynchronize();

				CFL3Kernel <<<1, threadsPerBlock>>>(
					dev_result, 
					dev_dtval, dev_maxW, dev_TotalTime);
				hipDeviceSynchronize();
				errCpy = hipMemcpy(dt_h, dev_dtval, sizeof(double)* 1, hipMemcpyDeviceToHost);
				errCpy = hipMemcpy(TotalStep_h, dev_TotalTime, sizeof(double)* 1, hipMemcpyDeviceToHost);
				
				if((*TotalStep_h + *dt_h) >= tf[iter]){
					*dt_h =  tf[iter] - *TotalStep_h;
					schreiben = 1;
					errCpy = hipMemcpy(dev_dtval, dt_h, sizeof(double)* 1, hipMemcpyHostToDevice);

					iter++;
				}
				
				if(*TotalStep_h >= inflowTime[inflowCount] && inflowCount < (Iniflowlen-1) ){
					inflowCount++;
					
				}
				hipDeviceSynchronize();
				fprintf(stdout, "\r\t\tSimulation progress: %3.0f %%", (*TotalStep_h*0.1/TotalSim)*100);
				fflush(stdout);

			}



			CFL4Kernel <<<1, threadsPerBlock>>>(
				dev_result, 
				dev_dt, dev_dtval,dev_TotalTime,io);
			hipDeviceSynchronize();


			Flux8Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_Hpx, dev_Hpy,
				dev_Ppx, dev_Ppy,
				dev_FpE, dev_FpW,
				dev_GpN, dev_GpS,

				dev_apEW, dev_apFEW,
				dev_apSN, dev_apFSN, 
				dev_uE, dev_uW, 
				dev_uN, dev_uS,
				dev_u,
				dev_ux, dev_uy,
				
				dev_Detmin_avgEW, dev_Detmin_avgSN,
				dev_cval_avgEW, dev_cval_avgSN,
				
				dev_invJ11_avgEW, dev_invJ12_avgEW, dev_invJ13_avgEW,  
				dev_invJ21_avgEW, dev_invJ22_avgEW, dev_invJ23_avgEW,  
				dev_invJ31_avgEW, dev_invJ32_avgEW, dev_invJ33_avgEW,  
				
				dev_invJ11_avgSN, dev_invJ12_avgSN, dev_invJ13_avgSN,  
				dev_invJ21_avgSN, dev_invJ22_avgSN, dev_invJ23_avgSN,  
				dev_invJ31_avgSN, dev_invJ32_avgSN, dev_invJ33_avgSN,  
				
				dev_dudxE, dev_dvdxE, 
				dev_dudyE, dev_dvdyE,

				dev_dudxN, dev_dvdxN, 
				dev_dudyN, dev_dvdyN,
				
				dx, dy,

				nxd, nyd);
			hipDeviceSynchronize();

			Flux9Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_duxidxix, dev_dvetdxix,
				dev_duxidetx, dev_dvetdetx,
				dev_duxidxiy, dev_dvetdxiy,
				dev_duxidety, dev_dvetdety,
				
				dev_invJ11_avgEW, dev_invJ12_avgEW, dev_invJ13_avgEW,  
				dev_invJ21_avgEW, dev_invJ22_avgEW, dev_invJ23_avgEW,  
				dev_invJ31_avgEW, dev_invJ32_avgEW, dev_invJ33_avgEW,  
				
				dev_invJ11_avgSN, dev_invJ12_avgSN, dev_invJ13_avgSN,  
				dev_invJ21_avgSN, dev_invJ22_avgSN, dev_invJ23_avgSN,  
				dev_invJ31_avgSN, dev_invJ32_avgSN, dev_invJ33_avgSN,  
				
				dev_dudxE, dev_dvdxE, 
				dev_dudyE, dev_dvdyE,

				dev_dudxN, dev_dvdxN, 
				dev_dudyN, dev_dvdyN,

				nxd, nyd);
			hipDeviceSynchronize();


			Flux10Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_PDx, dev_PDy,

				dev_uE, dev_uW, 
				dev_uN, dev_uS,
				
				dev_Detmin_avgEW, dev_Detmin_avgSN,

				dev_duxidxix, dev_dvetdxix,
				dev_duxidetx, dev_dvetdetx,
				dev_duxidxiy, dev_dvetdxiy,
				dev_duxidety, dev_dvetdety,
				
				dev_invJ11_avgEW, dev_invJ12_avgEW,
				dev_invJ21_avgEW, dev_invJ22_avgEW, 

				dev_invJ11_avgSN, dev_invJ12_avgSN,   
				dev_invJ21_avgSN, dev_invJ22_avgSN, 

				nxd, nyd);
			hipDeviceSynchronize();

			Flux11Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_u, 
				dev_vex, dev_vey,
				dev_vexF, dev_veyF, 
				nxd, nyd);
			hipDeviceSynchronize();

			Flux12Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_w_wert, dev_w_wertF,
				dev_vex, dev_vey,
				dev_vexF, dev_veyF, 
				dev_svec, dev_cvalue,
				nxd, nyd);
			hipDeviceSynchronize();

			Flux13Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_w_wert, dev_w_wertF,
				dev_vex, dev_vey,
				dev_usw, dev_vel, 
				nxd, nyd);
			hipDeviceSynchronize();

			Flux14Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_w_wert, dev_w_wertF,
				dev_vex, dev_vey,
				dev_vexF, dev_veyF,
				dev_vexw, dev_veyw, 
				dev_usw, dev_vel, 
				dev_q_xi, dev_q_et,
				dev_q_xiF, dev_q_etF,
				dev_invJ11, dev_invJ12, dev_invJ13,
				dev_invJ21, dev_invJ22, dev_invJ23,
				nxd, nyd);
			hipDeviceSynchronize();

			Flux15Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_w_wert, dev_w_wertF,
				dev_vex, dev_vey,
				dev_vexF, dev_veyF,
				dev_q_xi, dev_q_et,
				dev_q_xiF, dev_q_etF,
				dev_J13dxi, dev_J23dxi, dev_J33dxi,
				dev_J13det, dev_J23det, dev_J33det,
				dev_Ac, dev_AcF,
				nxd, nyd);
			hipDeviceSynchronize();

			Flux16Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_Npress1, dev_Npress2, dev_NpressF,
				dev_Ac, dev_AcF,
				dev_Detmin, dev_cvalue,
				dev_u,
				nxd, nyd);
			hipDeviceSynchronize();

			Flux17Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_Npress1, dev_Npress2, dev_NpressF,
				dev_Ac, dev_AcF,
				dev_Detmin, dev_svec,
				dev_vex, dev_vey,
				dev_vexw, dev_veyw,
				dev_vexF, dev_veyF,
				dev_tande,
				dev_u, dev_s,
				Cd, N_R, varTheta,
				nxd, nyd);
			hipDeviceSynchronize();

			Flux18Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_v, dev_Detmin, 
				dev_Hpx, dev_Hpy,
				dev_Ppx, dev_Ppy,
				dev_PDx, dev_PDy,
				dev_s,
				dev_u, dev_uzero,
				dev_dt,
				dx, dy,
				N_R,
				nxd, nyd);
			hipDeviceSynchronize();

			Flux19Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_v, dev_Detmin, 
				dev_u, 
				nxd, nyd);
			hipDeviceSynchronize();

			Flux20Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_u, dev_utmp,
				nxd, nyd);
			hipDeviceSynchronize();

			Flux21Kernel <<<blocksPerGrid, threadsPerBlock>>>(
				dev_result, 
				dev_u, 
				nxd, nyd);
			hipDeviceSynchronize();
			
			if(io == 0)
			{
			
				Flux22Kernel <<<blocksPerGrid, threadsPerBlock>>>(
					dev_result, 
					dev_u, dev_uone,
					nxd, nyd);
				hipDeviceSynchronize();

				Flux23Kernel <<<blocksPerGrid, threadsPerBlock>>>(
					dev_result, 
					dev_u, dev_uzero,
					dev_usxnew, dev_ufxnew, 
					dev_usxold, dev_ufxold, 
					dev_usynew, dev_ufynew, 
					dev_usyold, dev_ufyold, 
					nxd, nyd);
				hipDeviceSynchronize();

				Flux24Kernel <<<blocksPerGrid, threadsPerBlock>>>(
					dev_result, 
					dev_u, dev_uone,
					dev_usxnew, dev_ufxnew, 
					dev_usxold, dev_ufxold, 
					dev_usynew, dev_ufynew, 
					dev_usyold, dev_ufyold, 
					nxd, nyd);
				hipDeviceSynchronize();
			
			}
			else{
				Flux25Kernel <<<blocksPerGrid, threadsPerBlock>>>(
					dev_result, 
					dev_u, dev_utwo, 
					nxd, nyd);
				hipDeviceSynchronize();

				Flux26Kernel <<<blocksPerGrid, threadsPerBlock>>>(
					dev_result, 
					dev_u, dev_uone,
					dev_usxnew, dev_ufxnew, 
					dev_usxold, dev_ufxold, 
					dev_usynew, dev_ufynew, 
					dev_usyold, dev_ufyold, 
					nxd, nyd);
				hipDeviceSynchronize();

				Flux27Kernel <<<blocksPerGrid, threadsPerBlock>>>(
					dev_result, 
					dev_u, dev_utwo,
					dev_usxnew, dev_ufxnew, 
					dev_usxold, dev_ufxold, 
					dev_usynew, dev_ufynew, 
					dev_usyold, dev_ufyold, 
					nxd, nyd);
				hipDeviceSynchronize();

				Flux28Kernel <<<blocksPerGrid, threadsPerBlock>>>(
					dev_result, 
					dev_u, dev_utwo, 
					nxd, nyd);
				hipDeviceSynchronize();
			}


				if(io)
				{
					if(schreiben==1){
	
						errCpy = hipMemcpy(resultHs_h, &dev_u[0 * nyd * nxd], sizeof(double)* arraySize, hipMemcpyDeviceToHost);
						errCpy = hipMemcpy(resultHf_h, &dev_u[3 * nyd * nxd], sizeof(double)* arraySize, hipMemcpyDeviceToHost);
						errCpy = hipMemcpy(resultUs_h, &dev_u[1 * nyd * nxd], sizeof(double)* arraySize, hipMemcpyDeviceToHost);
						errCpy = hipMemcpy(resultVs_h, &dev_u[2 * nyd * nxd], sizeof(double)* arraySize, hipMemcpyDeviceToHost);
						errCpy = hipMemcpy(resultUf_h, &dev_u[4 * nyd * nxd], sizeof(double)* arraySize, hipMemcpyDeviceToHost);
						errCpy = hipMemcpy(resultVf_h, &dev_u[5 * nyd * nxd], sizeof(double)* arraySize, hipMemcpyDeviceToHost);
						errCpy = hipMemcpy(resultphi_h,&dev_u[6 * nyd * nxd], sizeof(double)* arraySize, hipMemcpyDeviceToHost);
					
					
						char outfile_Web[100]; sprintf(outfile_Web,"./result2Pb/%03d.dat",iter);

						outtime[iter-1] = *TotalStep_h;
						FILE *fpTmp;
						fpTmp=fopen("./result2Pb/Time.dat", "w");
						for (int nn=0;nn<(iter);nn++){
							fprintf(fpTmp, "%20.4f", outtime[nn]);
						}
						fclose(fpTmp);


						FILE  *fpout;

						fpout=fopen(outfile_Web, "w");
						fprintf(fpout, "VARIABLES = \"H\", \"phi\", \"Us\", \"Uf\", \"Vs\", \"Vf\"\n ");
						for (int i=MD;i<nxd-MD;i++) {
							for (int j=MD;j<nyd-MD;j++) {
								Htmp  = 0.1*(resultHs_h[j * nxd + i] + resultHf_h[j * nxd + i]);
								hstmp = resultHs_h[j * nxd + i];
								hftmp = resultHf_h[j * nxd + i];

								if (hstmp > 0.00001){
									vstmp = resultVs_h[j * nxd + i]/resultHs_h[j * nxd + i];
									ustmp = resultUs_h[j * nxd + i]/resultHs_h[j * nxd + i];
									phitmp = resultphi_h[j * nxd + i];
								}else{
									vstmp = 0.0;
									ustmp = 0.0;
									phitmp = 0.0;
								}

								if (hftmp > 0.00001){
									vftmp = resultVf_h[j * nxd + i]/resultHf_h[j * nxd + i];
									uftmp = resultUf_h[j * nxd + i]/resultHf_h[j * nxd + i];
								}else{
									vftmp = 0.0;
									uftmp = 0.0;
								}

								fprintf(fpout, "%10.4f\t%10.4f\t%10.4f\t%10.4f\t%10.4f\t%10.4f\n",Htmp,phitmp,ustmp,uftmp,vstmp,vftmp);
							} 
						}
						fclose(fpout);

						if(iter == outsteplen){
							nstop = 1;
							// Totalnt = nt;
						}
						// }
						schreiben = 0;
					}
				}

			}
			
			hipDeviceSynchronize();

		}

	if (hipPeekAtLastError() != hipSuccess) 
	{
			cout << hipGetErrorString(hipPeekAtLastError()) << endl;
	}

	errCpy = hipMemcpy(TotalStep_h, dev_TotalTime, sizeof(double)* 1, hipMemcpyDeviceToHost);
	cout << "\nTotal time : " << *TotalStep_h/10 << " sec  ";
	fprintf(stdout, "\nTotal number of steps: %d\n", nt);

	if(errCpy != hipSuccess){
		printf("\nError cuda Memory copy : %d\n",errCpy);
	}
}


void RunKernels::memoryMalloc()
{
	hipError_t errdevice ;

	errdevice = hipMalloc((void **)&dev_topo, sizeof(double) * arraySize); 
	errdevice = hipMalloc((void **)&dev_depth, sizeof(double) * arraySize); 
	errdevice = hipMalloc((void **)&dev_result, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_bfkt, sizeof(double) * arraySize * 3);

	errdevice = hipMalloc((void **)&dev_posx, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_posy, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_dxdxi11, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dxdxi12, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dxdxi21, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dxdxi22, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_dbdx  , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dbdy  , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_cvalue, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_svec  , sizeof(double) * arraySize * 2);
	errdevice = hipMalloc((void **)&dev_Jacb31, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_Jacb32, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_dettmp, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_Detmin, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_i_ddxi11, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_i_ddxi12, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_i_ddxi21, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_i_ddxi22, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_invJ11, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ12, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ13, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_invJ21, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ22, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ23, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_invJ31, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ32, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ33, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_u     , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_uzero , sizeof(double) * arraySize * 7);

	errdevice = hipMalloc((void **)&dev_Hpx   , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_Hpy   , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_Ppx   , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_Ppy   , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_PDx   , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_PDy   , sizeof(double) * arraySize * 7);

	errdevice = hipMalloc((void **)&dev_ux    , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_uy    , sizeof(double) * arraySize * 7);

	errdevice = hipMalloc((void **)&dev_dux   , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_duy   , sizeof(double) * arraySize * 7);

	errdevice = hipMalloc((void **)&dev_t1x   , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_t2x   , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_t1y   , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_t2y   , sizeof(double) * arraySize * 7);
	
	errdevice = hipMalloc((void **)&dev_sgnAx , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_sgnBx , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_sgnAy , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_sgnBy , sizeof(double) * arraySize * 7);

	errdevice = hipMalloc((void **)&dev_uE , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_uW , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_uN , sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_uS , sizeof(double) * arraySize * 7);

	errdevice = hipMalloc((void **)&dev_tande , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_J13dxi , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J23dxi , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J33dxi , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J13det , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J23det , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J33det , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_apEW  , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_apSN  , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_apFEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_apFSN , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_dxdxi11_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dxdxi21_avgEW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_dxdxi12_avgSN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dxdxi22_avgSN , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_J13dxi_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J23dxi_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J33dxi_avgEW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_J13det_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J23det_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J33det_avgEW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_J13dxi_avgSN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J23dxi_avgSN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J33dxi_avgSN , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_J13det_avgSN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J23det_avgSN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_J33det_avgSN , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_invJ11_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ12_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ13_avgEW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_invJ21_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ22_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ23_avgEW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_invJ31_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ32_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ33_avgEW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_invJ11_avgSN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ12_avgSN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ13_avgSN , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_invJ21_avgSN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ22_avgSN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ23_avgSN , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_invJ31_avgSN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ32_avgSN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_invJ33_avgSN , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_Detmin_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_Detmin_avgSN , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_cval_avgEW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_cval_avgSN , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_svec_avgEW , sizeof(double) * arraySize * 2);
	errdevice = hipMalloc((void **)&dev_svec_avgSN , sizeof(double) * arraySize * 2);

	errdevice = hipMalloc((void **)&dev_vexE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_vexW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_veyE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_veyW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_w_wertE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_w_wertW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_vexFE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_vexFW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_veyFE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_veyFW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_w_wertFE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_w_wertFW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_vexN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_vexS , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_veyN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_veyS , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_w_wertN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_w_wertS , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_vexFN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_vexFS , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_veyFN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_veyFS , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_w_wertFN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_w_wertFS , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_q_xiE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_etE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_xiW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_etW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_q_xiFE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_etFE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_xiFW , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_etFW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_NpressFE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_NpressFW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_M11EW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_q_xiN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_etN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_xiS , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_etS , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_q_xiFN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_etFN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_xiFS , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_etFS , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_NpressFN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_NpressFS , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_M22SN , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_apE  , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_apW  , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_apFE , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_apFW , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_apN  , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_apS  , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_apFN , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_apFS , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_em_x , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_em_y , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_em_Fx, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_em_Fy, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_FpE, sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_FpW, sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_GpN, sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_GpS, sizeof(double) * arraySize * 7);

	errdevice = hipMalloc((void **)&dev_czw1x , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_czw2x , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_czwF1x, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_czwF2x, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_czw1y , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_czw2y , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_czwF1y, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_czwF2y, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_em_valS, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_em_valF, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_Val    , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_dudxE, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dvdxE, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dudyE, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dvdyE, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_dudxN, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dvdxN, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dudyN, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dvdyN, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_duxidxix, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dvetdxix, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_duxidetx, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dvetdetx, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_duxidxiy, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dvetdxiy, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_duxidety, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_dvetdety, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_vex    , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_vey    , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_vexF   , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_veyF   , sizeof(double) * arraySize);
	
	errdevice = hipMalloc((void **)&dev_w_wert , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_w_wertF, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_usw    , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_vel    , sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_vexw   , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_veyw   , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_xi   , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_et   , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_xiF  , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_q_etF  , sizeof(double) * arraySize);
	
	errdevice = hipMalloc((void **)&dev_Ac     , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_AcF    , sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_Npress1, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_Npress2, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_NpressF, sizeof(double) * arraySize);
	
	errdevice = hipMalloc((void **)&dev_s, sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_v, sizeof(double) * arraySize * 7);

	errdevice = hipMalloc((void **)&dev_uone, sizeof(double) * arraySize * 7);
	errdevice = hipMalloc((void **)&dev_utwo, sizeof(double) * arraySize * 7);

	errdevice = hipMalloc((void **)&dev_usxnew, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_ufxnew, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_usxold, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_ufxold, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_usynew, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_ufynew, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_usyold, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_ufyold, sizeof(double) * arraySize);

	errdevice = hipMalloc((void **)&dev_utmp, sizeof(double) * arraySize * 7);

	errdevice = hipMalloc((void **)&dev_waveSpeed, sizeof(double) * arraySize);
	errdevice = hipMalloc((void **)&dev_max, sizeof(double) * 256);
	errdevice = hipMalloc((void **)&dev_maxW, sizeof(double) * 1);
	errdevice = hipMalloc((void **)&dev_TotalTime, sizeof(double) * 1);
	errdevice = hipMalloc((void **)&dev_dt, sizeof(double) * 1);
	errdevice = hipMalloc((void **)&dev_dtval, sizeof(double) * 1);

	errdevice = hipMalloc((void **)&dev_inflow, sizeof(double) * (inflowSize * 3));
	errdevice = hipMalloc((void **)&dev_loc, sizeof(double) * (locflowlen * 3));

	errdevice = hipMalloc((void **)&dev_dire, sizeof(int) * (locflowlen * 3));

	if(errdevice != hipSuccess){
		printf("\nCould not allocate Device memory : %d\n",errdevice);
	}

}

void RunKernels::freeMemory()
{
	hipFree(dev_topo);
	hipFree(dev_depth);
	
	hipFree(dev_result);

	// hipFree(dev_bfkt);
	hipFree(dev_posx);
	hipFree(dev_posy);

	hipFree(dev_dxdxi11);
	hipFree(dev_dxdxi12);
	hipFree(dev_dxdxi21);
	hipFree(dev_dxdxi22);

	hipFree(dev_dbdx);
	hipFree(dev_dbdy);
	hipFree(dev_cvalue);

	hipFree(dev_svec);
	hipFree(dev_Jacb31);
	hipFree(dev_Jacb32);

	// hipFree(dev_dettmp);
	hipFree(dev_Detmin);

	// hipFree(dev_i_ddxi11);
	// hipFree(dev_i_ddxi12);
	// hipFree(dev_i_ddxi21);
	// hipFree(dev_i_ddxi22);

	hipFree(dev_invJ11);
	hipFree(dev_invJ12);
	hipFree(dev_invJ13);

	hipFree(dev_invJ21);
	hipFree(dev_invJ22);
	hipFree(dev_invJ23);

	hipFree(dev_invJ31);
	hipFree(dev_invJ32);
	hipFree(dev_invJ33);

	hipFree(dev_u);
	hipFree(dev_uzero);

	hipFree(dev_tande);

	hipFree(dev_J13dxi);
	hipFree(dev_J23dxi);
	hipFree(dev_J33dxi);
	hipFree(dev_J13det);
	hipFree(dev_J23det);
	hipFree(dev_J33det);

	hipFree(dev_Hpx);
	hipFree(dev_Hpy);
	hipFree(dev_Ppx);
	hipFree(dev_Ppy);
	hipFree(dev_PDx);
	hipFree(dev_PDy);
	hipFree(dev_ux);
	hipFree(dev_uy);

	hipFree(dev_apEW);
	hipFree(dev_apSN);
	hipFree(dev_apFEW);
	hipFree(dev_apFSN);

	hipFree(dev_dux);
	hipFree(dev_duy);

	hipFree(dev_t1x);
	hipFree(dev_t2x);
	hipFree(dev_t1y);
	hipFree(dev_t2y);

	hipFree(dev_sgnAx);
	hipFree(dev_sgnBx);
	hipFree(dev_sgnAy);
	hipFree(dev_sgnBy);

	hipFree(dev_dxdxi11_avgEW);
	hipFree(dev_dxdxi21_avgEW);

	hipFree(dev_dxdxi12_avgSN);
	hipFree(dev_dxdxi22_avgSN);

	hipFree(dev_J13dxi_avgEW);
	hipFree(dev_J23dxi_avgEW);
	hipFree(dev_J33dxi_avgEW);

	hipFree(dev_J13det_avgEW);
	hipFree(dev_J23det_avgEW);
	hipFree(dev_J33det_avgEW);

	hipFree(dev_J13dxi_avgSN);
	hipFree(dev_J23dxi_avgSN);
	hipFree(dev_J33dxi_avgSN);

	hipFree(dev_J13det_avgSN);
	hipFree(dev_J23det_avgSN);
	hipFree(dev_J33det_avgSN);

	hipFree(dev_invJ11_avgEW);
	hipFree(dev_invJ12_avgEW);
	hipFree(dev_invJ13_avgEW);

	hipFree(dev_invJ21_avgEW);
	hipFree(dev_invJ22_avgEW);
	hipFree(dev_invJ23_avgEW);

	hipFree(dev_invJ31_avgEW);
	hipFree(dev_invJ32_avgEW);
	hipFree(dev_invJ33_avgEW);

	hipFree(dev_invJ11_avgSN);
	hipFree(dev_invJ12_avgSN);
	hipFree(dev_invJ13_avgSN);

	hipFree(dev_invJ21_avgSN);
	hipFree(dev_invJ22_avgSN);
	hipFree(dev_invJ23_avgSN);

	hipFree(dev_invJ31_avgSN);
	hipFree(dev_invJ32_avgSN);
	hipFree(dev_invJ33_avgSN);

	hipFree(dev_Detmin_avgEW);
	hipFree(dev_Detmin_avgSN);

	hipFree(dev_cval_avgEW);
	hipFree(dev_cval_avgSN);

	hipFree(dev_svec_avgEW);
	hipFree(dev_svec_avgSN);

	hipFree(dev_uE);
	hipFree(dev_uW);
	hipFree(dev_uN);
	hipFree(dev_uS);

	hipFree(dev_vexE);
	hipFree(dev_vexW);
	hipFree(dev_veyE);
	hipFree(dev_veyW);

	hipFree(dev_w_wertE);
	hipFree(dev_w_wertW);

	hipFree(dev_vexFE);
	hipFree(dev_vexFW);
	hipFree(dev_veyFE);
	hipFree(dev_veyFW);

	hipFree(dev_w_wertFE);
	hipFree(dev_w_wertFW);

	hipFree(dev_vexN);
	hipFree(dev_vexS);
	hipFree(dev_veyN);
	hipFree(dev_veyS);

	hipFree(dev_w_wertFN);
	hipFree(dev_w_wertFS);

	hipFree(dev_q_xiE);
	hipFree(dev_q_etE);
	hipFree(dev_q_xiW);
	hipFree(dev_q_etW);

	hipFree(dev_q_xiFE);
	hipFree(dev_q_etFE);
	hipFree(dev_q_xiFW);
	hipFree(dev_q_etFW);

	hipFree(dev_NpressFE);
	hipFree(dev_NpressFW);

	hipFree(dev_M11EW);

	hipFree(dev_q_xiN);
	hipFree(dev_q_etN);
	hipFree(dev_q_xiS);
	hipFree(dev_q_etS);

	hipFree(dev_q_xiFN);
	hipFree(dev_q_etFN);
	hipFree(dev_q_xiFS);
	hipFree(dev_q_etFS);

	hipFree(dev_NpressFN);
	hipFree(dev_NpressFS);

	hipFree(dev_M22SN);

	hipFree(dev_apE);
	hipFree(dev_apW);
	hipFree(dev_apFE);
	hipFree(dev_apFW);

	hipFree(dev_apN);
	hipFree(dev_apS);
	hipFree(dev_apFN);
	hipFree(dev_apFS);

	hipFree(dev_em_x);
	hipFree(dev_em_y);
	hipFree(dev_em_Fx);
	hipFree(dev_em_Fy);

	hipFree(dev_FpE);
	hipFree(dev_FpW);
	hipFree(dev_GpN);
	hipFree(dev_GpS);

	hipFree(dev_czw1x);
	hipFree(dev_czw2x);
	hipFree(dev_czwF1x);
	hipFree(dev_czwF2x);

	hipFree(dev_czw1y);
	hipFree(dev_czw2y);
	hipFree(dev_czwF1y);
	hipFree(dev_czwF2y);

	hipFree(dev_em_valS);
	hipFree(dev_em_valF);
	hipFree(dev_Val);

	hipFree(dev_dudxE);
	hipFree(dev_dvdxE);
	hipFree(dev_dudyE);
	hipFree(dev_dvdyE);

	hipFree(dev_dudxN);
	hipFree(dev_dvdxN);
	hipFree(dev_dudyN);
	hipFree(dev_dvdyN);

	hipFree(dev_duxidxix);
	hipFree(dev_dvetdxix);
	hipFree(dev_duxidetx);
	hipFree(dev_dvetdetx);

	hipFree(dev_duxidxiy);
	hipFree(dev_dvetdxiy);
	hipFree(dev_duxidety);
	hipFree(dev_dvetdety);

	hipFree(dev_vex);
	hipFree(dev_vey);
	hipFree(dev_vexF);
	hipFree(dev_veyF);

	hipFree(dev_w_wert);
	hipFree(dev_w_wertF);
	hipFree(dev_usw);
	hipFree(dev_vel);

	hipFree(dev_vexw);
	hipFree(dev_veyw);
	hipFree(dev_q_xi);
	hipFree(dev_q_et);
	hipFree(dev_q_xiF);
	hipFree(dev_q_etF);

	hipFree(dev_Ac);
	hipFree(dev_AcF);
	hipFree(dev_Npress1);
	hipFree(dev_Npress2);
	hipFree(dev_NpressF);

	hipFree(dev_s);
	hipFree(dev_v);

	hipFree(dev_uone);
	hipFree(dev_utwo);

	hipFree(dev_usxnew);
	hipFree(dev_ufxnew);
	hipFree(dev_usxold);
	hipFree(dev_ufxold);

	hipFree(dev_utmp);

	hipFree(dev_inflow);
	hipFree(dev_loc);
	hipFree(dev_dire);

	hipFree(dev_waveSpeed);
	hipFree(dev_max);
	hipFree(dev_maxW);
	hipFree(dev_TotalTime);
	hipFree(dev_dt);

	
}